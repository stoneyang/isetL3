
#include <hip/hip_runtime.h>
#include <stdint.h>

// Define parameters
#define cfa_size      4
#define num_filter    5
#define patch_size    9
#define border_size   4
#define image_width   720
#define image_height  1280
#define voltage_max   0.9734
#define lum_list_size 20
#define num_out       3
#define low           0.95
#define high          1.15

/* Cuda function - L3Render

  Compute mean for each channel

  Inputs:
    out_image       - pre-allocated space for output (xyz) image
    image           - original image
    cfa             - cfa pattern, should be 0-indexed
    lum_list        - luminance list
    sat_list        - saturation list
    flat_filters    - filters for flat regions
    texture_filters - filters for texture regions
*/
__global__
void L3Render(float* const out_image,
                 float  const * const image,
                 unsigned short const * const cfa,
                 float  const * const lum_list,
                 float  const * const sat_list,
                 float  const * const flat_filters,
                 float  const * const texture_filters,
                 float  const * const flat_threshold_list)
{
    // Find pixel position
    const int row = blockIdx.x;
    const int col = threadIdx.x;
    
    // Check pixel range
    if ((row < border_size) || 
        (row >= image_height - border_size) ||
        (col < border_size) ||
        (col >= image_width - border_size))
        return;
    
    // Compute patch type
    const unsigned short patch_type[] = {row % cfa_size, col % cfa_size};
    
    // Compute mean for each channel
    float channel_mean[num_filter] = {0.0};
    unsigned short channel_count[num_filter] = {0};
    unsigned short cfa_index[patch_size * patch_size];

    for (short ii = -border_size; ii <= border_size; ii++){
        for (short jj = -border_size; jj <= border_size; jj++){
            unsigned short index = ii + border_size + (jj + border_size) * patch_size;
            cfa_index[index] = ((row + ii) % cfa_size) +
                               ((col + jj) % cfa_size) * cfa_size;
            channel_count[cfa[cfa_index[index]]] += 1;
            channel_mean[cfa[cfa_index[index]]] += image[row + ii + (col + jj) * image_height];
        }
    }

    // Compute channel mean and luminance
    float lum_mean = 0;
    size_t pixel_index = row + col * image_height;
    for (int ii = 0; ii < num_filter; ii++) {
        channel_mean[ii] /= channel_count[ii];
        lum_mean += channel_mean[ii];
    }
    lum_mean /= num_filter;
    
    // Convert luminance to luminance index
    // Binary search will be faster, but, we just use linear search for simplicity
    unsigned short lum_index = lum_list_size - 1;
    for (int ii = 0; ii < lum_list_size; ii++) {
        if (lum_mean < lum_list[ii]) {
            lum_index = ii;
            break;
        }
    }
    
    // Compute saturation type
    unsigned short sat_type = 0; // sat_type is the encoded saturation type
    unsigned short sat_index;    // sat_index is the number found with sat_list
    const unsigned short sat_list_size = (1 << num_filter);
    for (int ii = num_filter - 1; ii >= 0; ii --)
        sat_type = sat_type << 1 + (channel_mean[ii] > voltage_max);
    
    const float *cur_sat_list = sat_list + (patch_type[1] * cfa_size + patch_type[0]) * sat_list_size;
    sat_index = cur_sat_list[sat_type];
    
    // Find nearest sat_type for missing ones
    if (sat_index == 0){
        float min_cost = 10000; // Init min cost to some arbitrarily large value
        for (int ii = 0; ii < sat_list_size; ii++) {
            if (cur_sat_list[ii] == 0) continue;
            // compute cost
            float cur_cost = 0;
            for (int jj = 0; jj < num_filter; jj++) {
                if (((ii ^ sat_type) & (1 << jj)) > 0) {
                    float dist_to_max = channel_mean[jj] - voltage_max;
                    cur_cost += (dist_to_max > 0 ? dist_to_max : -dist_to_max);
                }
            }
            if (cur_cost < min_cost) {
                min_cost = cur_cost;
                sat_index = cur_sat_list[ii];
            }
        }
    }
    sat_index--; // make sat_index 0-indexed

    // Compute image contrast
    // Assume image_contrast array has been allocated as zeros
    float image_contrast = 0;
    for (int ii = -border_size; ii <= border_size; ii++){
        for (int jj = -border_size; jj <= border_size; jj++){
            unsigned short index = ii + border_size + (jj + border_size) * patch_size;
            size_t cur_pixel_index = row + ii + (col + jj) * image_height;
            float dist_to_mean = image[cur_pixel_index] - channel_mean[cfa[cfa_index[index]]];
            if (dist_to_mean > 0)
                image_contrast += dist_to_mean;
            else
                image_contrast -= dist_to_mean;
        }
    }
    image_contrast /= num_filter;
    
    // Determine flat or texture
    int threshold_index  = ((sat_index * lum_list_size + lum_index) 
                            * cfa_size + patch_type[1]) * cfa_size + patch_type[0];
    float flat_threshold = flat_threshold_list[threshold_index];
    
    // Apply filter to patch
    const float *filter;
    if (image_contrast < flat_threshold * low) { // flat region
        filter = flat_filters + threshold_index * num_out * patch_size * patch_size;
        for (int ii = -border_size; ii <= border_size; ii++){
            for (int jj = -border_size; jj <= border_size; jj++){
                unsigned short index = (ii + border_size + (jj + border_size) * patch_size)*3;
                size_t cur_pixel_index = row + ii + (col + jj) * image_height;
                out_image[pixel_index] += image[cur_pixel_index] * filter[index];
                out_image[pixel_index + image_width * image_height] += image[cur_pixel_index] * filter[index + 1];
                out_image[pixel_index + 2 * image_width * image_height] += image[cur_pixel_index] * filter[index + 2];
            }
        }
    }
    else if (image_contrast > flat_threshold * high) { // texture region
        filter = texture_filters + threshold_index * num_out * patch_size * patch_size;
        for (int ii = -border_size; ii <= border_size; ii++){
            for (int jj = -border_size; jj <= border_size; jj++){
                unsigned short index = (ii + border_size + (jj + border_size) * patch_size)*3;
                size_t cur_pixel_index = row + ii + (col + jj) * image_height;
                out_image[pixel_index] += image[cur_pixel_index] * filter[index];
                out_image[pixel_index + image_width * image_height] += image[cur_pixel_index] * filter[index + 1];
                out_image[pixel_index + 2 * image_width * image_height] += image[cur_pixel_index] * filter[index + 2];
            }
        }
    } 
    else { // transition region
        const float weights = (image_contrast / flat_threshold - low) / (high - low);
        filter = flat_filters + threshold_index * num_out * patch_size * patch_size;
        const float* filter_texture = texture_filters + threshold_index * num_out * patch_size * patch_size;
        for (int ii = -border_size; ii <= border_size; ii++){
            for (int jj = -border_size; jj <= border_size; jj++){
                unsigned short index = (ii + border_size + (jj + border_size) * patch_size) * 3;
                size_t cur_pixel_index = row + ii + (col + jj) * image_height;
                out_image[pixel_index] += image[cur_pixel_index] * (filter[index] * weights + filter_texture[index] * (1 - weights));
                out_image[pixel_index + image_width * image_height] += image[cur_pixel_index] * (filter[index + 1] * weights + filter_texture[index + 1]*(1-weights));
                out_image[pixel_index + 2 * image_width * image_height] += image[cur_pixel_index] * (filter[index + 2]*weights + filter_texture[index+2]*(1-weights));
            }
        }
    }
}